#include "hip/hip_runtime.h"
#include "probability.h"
#include <math.h>

__inline__
__device__
uint32_t randint(uint32_t a, uint32_t b)
{
	return (rand() % (b - a - 1)) + a;
}

__inline__
__device__
uint64_t factorial(uint64_t n)
{
	return tgamma(n + 1);
}

__inline__
__device__
uint64_t binomial_coef(uint64_t n, uint64_t k)
{
	return factorial(n) / (factorial(k) * factorial(n - k));
}

__inline__
__device__
uint8_t binomial_dist(uint32_t k, uint32_t n, float p)
{
	uint32_t prob = 100 * (float)binomial_coef(k, n) * pow(p, k) * pow(1 - p, n - k);
	return randint(0, 100) <= prob;		
}
